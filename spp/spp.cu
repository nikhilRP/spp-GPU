/******************************************************************************
  spp.cpp

  (c) 2014 - Nikhil R Podduturi
  Cherry Lab, Stanford University

  Licensed under the GNU General Public License 2.0 license.
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/io/fmindex/fmindex.h>

#include "alignment.h"

void crcInit();

using namespace nvbio;
using namespace spp;

int main(int argc, char* argv[])
{
    Timer timer;
    timer.start();

    hipSetDeviceFlags( hipDeviceMapHost | hipDeviceLmemResizeToMax );

    crcInit();
    if (argc == 1 ||
        (argc == 2 && strcmp( argv[1], "--help" ) == 0) ||
        (argc == 2 && strcmp( argv[1], "-h" ) == 0))
    {
        log_info(stderr,"idr [options] -- bam1 bam2\n");
        exit(0);
    }

    //bool   debug      = false;
    int cuda_device  = -1;
    int device_count;
    hipGetDeviceCount(&device_count);
    log_verbose(stderr, "  cuda devices : %d\n", device_count);

    // inspect and select cuda devices
    if (device_count)
    {
        if (cuda_device == -1)
        {
            int            best_device = 0;
            hipDeviceProp_t best_device_prop;
            hipGetDeviceProperties( &best_device_prop, best_device );
            for (int device = 0; device < device_count; ++device)
            {
                hipDeviceProp_t device_prop;
                hipGetDeviceProperties( &device_prop, device );
                if (device_prop.major >= best_device_prop.major &&
                    device_prop.minor >= best_device_prop.minor)
                {
                    best_device_prop = device_prop;
                    best_device      = device;
                }
            }
            cuda_device = best_device;
        }
        log_verbose(stderr, "  chosen device %d\n", cuda_device);
        {
            hipDeviceProp_t device_prop;
            hipGetDeviceProperties( &device_prop, cuda_device );
            log_verbose(stderr, "    device name        : %s\n", device_prop.name);
            log_verbose(stderr, "    compute capability : %d.%d\n", device_prop.major, device_prop.minor);
        }
        hipSetDevice( cuda_device );
    }

    int arg = 1;
    try
    {
        if (argc == arg + 2)
        {
            const char *aln_file_nameL = argv[arg];
            const char *aln_file_nameR = argv[arg+1];

            SharedPointer<AlignmentStream> aln_streamL = SharedPointer<AlignmentStream>( open_alignment_file( aln_file_nameL ) );
            SharedPointer<AlignmentStream> aln_streamR = SharedPointer<AlignmentStream>( open_alignment_file( aln_file_nameR ) );

            printf("I'm here");

            if (aln_streamL == NULL || aln_streamL->is_ok() == false)
            {
                log_error(stderr, "failed opening \"%s\"\n", aln_file_nameL);
                exit(1);
            }

            if (aln_streamR == NULL || aln_streamR->is_ok() == false)
            {
                log_error(stderr, "failed opening \"%s\"\n", aln_file_nameR);
                exit(1);
            }

            const uint32 BATCH_SIZE = 500000;
            std::vector<Alignment> batchL( BATCH_SIZE );
            std::vector<Alignment> batchR( BATCH_SIZE );
        }
    }
    catch (nvbio::cuda_error e)
    {
        log_error(stderr, "caught a nvbio::cuda_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::bad_alloc e)
    {
        log_error(stderr, "caught a nvbio::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::logic_error e)
    {
        log_error(stderr, "caught a nvbio::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::runtime_error e)
    {
        log_error(stderr, "caught a nvbio::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::bad_alloc e)
    {
        log_error(stderr, "caught a std::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::logic_error &e)
    {
        log_error(stderr, "caught a std::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::runtime_error e)
    {
        log_error(stderr, "caught a std::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (...)
    {
        log_error(stderr, "caught an unknown exception!\n");
    }

    return 0;
}
